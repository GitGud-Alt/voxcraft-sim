#include "hip/hip_runtime.h"
#include "VX3_VoxelGroup.h"

#include "VX3_Voxel.h"
#include "VX3_VoxelyzeKernel.cuh"

__device__ VX3_VoxelGroup::VX3_VoxelGroup(VX3_VoxelyzeKernel *k) { d_kernel = k; }

__device__ void VX3_VoxelGroup::deviceInit() {
    d_surface_voxels.clear();
    d_voxels.clear();
}

__device__ void VX3_VoxelGroup::switchAllVoxelsTo(VX3_VoxelGroup *group) {
    if (group == this)
        return;
    for (int i = 0; i < d_voxels.size(); i++) {
        if (d_voxels[i]->d_group == NULL) {
            d_voxels[i]->d_group = group;
        } else if (d_voxels[i]->d_group != NULL && d_voxels[i]->d_group != group) {
            d_voxels[i]->d_group->removed = true;
            d_voxels[i]->d_group = group;
        } else {
            // d_group is already group.
        }
    }
}

__device__ VX3_Vec3D<int> VX3_VoxelGroup::moveGroupPosition(VX3_Vec3D<int> from, linkDirection dir, int step) {
    VX3_Vec3D<int> to = from;
    switch (dir) {
    case X_POS:
        to.x += step;
        break;
    case X_NEG:
        to.x -= step;
        break;
    case Y_POS:
        to.y += step;
        break;
    case Y_NEG:
        to.y -= step;
        break;
    case Z_POS:
        to.z += step;
        break;
    case Z_NEG:
        to.z -= step;
        break;
    default:
        printf("ERROR in moveGroupPosition.\n");
    }
    return to;
}

__device__ void VX3_VoxelGroup::updateGroup() {
    if (removed)
        return;
    if (lastBuildTime == d_kernel->currentTime)
        return;
    lastBuildTime = d_kernel->currentTime;

    VX3_Voxel *voxel = d_voxels[0];
    int min_x, min_y, min_z, max_x, max_y, max_z;
    min_x = 0;
    min_y = 0;
    min_z = 0;
    max_x = 0;
    max_y = 0;
    max_z = 0;

    // First set *voxel as origin, and negative number is allowed.
    // After everything is mapped, change origin to (min_x, min_y, min_z).
    voxel->groupPosition = VX3_Vec3D<int>(0, 0, 0);

    // Rebuild a map of Group
    //// BF Search for all neighbors
    VX3_dDictionary<VX3_Voxel *, int> BFS_visited;
    VX3_dQueue<VX3_Voxel *> BFS_Queue;
    VX3_dVector<VX3_Voxel *> BFS_result;

    BFS_result.push_back(voxel);
    BFS_visited.set(voxel, 1);
    BFS_Queue.push_back(voxel);

    while (!BFS_Queue.isEmpty()) {
        VX3_Voxel *v = BFS_Queue.pop_front();
        for (int i = 0; i < 6; i++) {
            VX3_Voxel *neighbor = v->adjacentVoxel((linkDirection)i);
            if (neighbor) {
                if (BFS_visited.get(neighbor) == -1) {
                    neighbor->groupPosition = moveGroupPosition(v->groupPosition, (linkDirection)i);
                    // Set all connected voxels' d_group to this
                    neighbor->d_group->switchAllVoxelsTo(this);

                    BFS_result.push_back(neighbor);
                    BFS_visited.set(neighbor, 1);
                    BFS_Queue.push_back(neighbor);
                    // update the min and max for later use
                    if (neighbor->groupPosition.x < min_x) {
                        min_x = neighbor->groupPosition.x;
                    }
                    if (neighbor->groupPosition.x > max_x) {
                        max_x = neighbor->groupPosition.x;
                    }
                    if (neighbor->groupPosition.y < min_y) {
                        min_y = neighbor->groupPosition.y;
                    }
                    if (neighbor->groupPosition.y > max_y) {
                        max_y = neighbor->groupPosition.y;
                    }
                    if (neighbor->groupPosition.z < min_z) {
                        min_z = neighbor->groupPosition.z;
                    }
                    if (neighbor->groupPosition.z > max_z) {
                        max_z = neighbor->groupPosition.z;
                    }
                } else {
                    // check the group position to make sure the connection is right
                    if (neighbor->groupPosition != moveGroupPosition(v->groupPosition, (linkDirection)i)) {
                        if (v->links[i])
                            v->links[i]->detach(); // TODO: there are potential racing conditions here.
                    }
                }
            }
        }
    }

    //// Allocate memory for the map
    dim_x = max_x - min_x + 1;
    dim_y = max_y - min_y + 1;
    dim_z = max_z - min_z + 1;
    if (buffer_size_group_map < dim_x * dim_y * dim_z) { // size of group map exceed the buffer size
        if (buffer_size_group_map == 0) {
            buffer_size_group_map = (dim_x * dim_y * dim_z >= 16) ? (dim_x * dim_y * dim_z * 2) : 32; // by default, allocate 10, so no need to go through 2,4,8,16,32
        } else {
            buffer_size_group_map = dim_x * dim_y * dim_z * 2; // double the size
        }
        if (d_group_map) {
            free(d_group_map);
            d_group_map = NULL;
        }
        d_group_map = (VX3_Voxel **)malloc(buffer_size_group_map * sizeof(VX3_Voxel *));
        if (!d_group_map) {
            printf("Out of Memory: d_group_map.\n");
        }
    }
    memset(d_group_map, 0, dim_x * dim_y * dim_z * sizeof(VX3_Voxel *)); // only use this much of the buffer
    d_voxels.clear();
    d_surface_voxels.clear();
    for (int i = 0; i < BFS_result.size(); i++) {
        VX3_Voxel *v = BFS_result[i];
        v->groupPosition.x -= min_x;
        v->groupPosition.y -= min_y;
        v->groupPosition.z -= min_z;

        // sam:
        bool absorb = false;
        if (d_kernel->keepJustOneIfManyHaveSameGroupPosition) {

            for (int j = 0; j < d_voxels.size(); j++) {
                if (v->groupPosition == d_voxels[j]->groupPosition) {
                    absorb = true;

                    // Option 1: delete it
                    // v->removed = true;

                    // Option 2: just break off
                    v->d_group = new VX3_VoxelGroup(d_kernel);
                    d_kernel->d_voxel_to_update_group.push_back(v);
                    v->d_group->d_voxels.push_back(v);
                    d_kernel->d_voxelgroups.push_back(v->d_group);

                    // either way, delete all the links
                    for (int k = 0; k < 6; k++) { // check links in all direction
                        if (v->links[k]) {
                            v->links[k]->removed = true;
                            v->adjacentVoxel((linkDirection)k)->links[oppositeDirection(k)] = NULL;
                            v->links[k] = NULL;
                        }
                    }
                }
            }
        }
        if (!absorb) { // sam
            int offset = to1D(v->groupPosition, VX3_Vec3D<int>(dim_x, dim_y, dim_z));
            d_group_map[offset] = v;
            d_voxels.push_back(v);
            // If any link is NULL => is surface voxel
            if (!(v->links[0] && v->links[1] && v->links[2] && v->links[3] && v->links[4] && v->links[5])) {
                d_surface_voxels.push_back(v);
            }
        } // sam
    }
}

////////////////////////////////////////////////////////////////////
// NOTE: There is another racing condition that I have not catched yet:
//  suppose there are three groups, it is compatible for any two of them. But consider all three groups, they are not compatible.
//  e.g.:   |    |      Three verticle bars like this.
//          |  | |      If watchDistance is large enough, this could happen.
//             |
////////////////////////////////////////////////////////////////////
__device__ bool VX3_VoxelGroup::isCompatible(VX3_Voxel *voxel_host, VX3_Voxel *voxel_remote, int *ret_linkdir_1, int *ret_linkdir_2) {
    if (voxel_host->d_group != this) {
        printf("This method should be call from voxel_host's d_group.\n"); // here?
        return false;
    }
    // Given two voxels, determine the best way to attach them.
    VX3_Vec3D<int> offset_of_link = VX3_Vec3D<int>(0, 0, 0);
    int potential_link_1, potential_link_2;
    VX3_Quat3D<double> relativeRotation = voxel_remote->orientation().Conjugate() * voxel_host->orientation();
    bool hasSingleton = false;

    if (true) { // Rotate singleton and small bar to align for attachment
        bool voxel_remote_singleton, voxel_remote_smallbar;
        int voxel_remote_direction;
        voxel_remote->isSingletonOrSmallBar(&voxel_remote_singleton, &voxel_remote_smallbar, &voxel_remote_direction);
        bool voxel_host_singleton, voxel_host_smallbar;
        int voxel_host_direction;
        voxel_host->isSingletonOrSmallBar(&voxel_host_singleton, &voxel_host_smallbar, &voxel_host_direction);
        if (voxel_remote_singleton || voxel_remote_smallbar || voxel_host_singleton || voxel_host_smallbar) {
            if (atomicCAS(&d_kernel->mutexRotateSingleton, 0, 1) == 0) {
                if (voxel_remote_singleton) { // remote has no link. so rotate the remote
                    voxel_remote->changeOrientationTo(voxel_host->orient);
                } else if (voxel_host_singleton) { // voxel host has no link, so rotate the host
                    voxel_host->changeOrientationTo(voxel_remote->orient);
                } else if (voxel_remote_smallbar) { // remote is a small bar  // they all have links, detach the one with less link
                    voxel_remote->adjacentVoxel((linkDirection)voxel_remote_direction)->changeOrientationTo(voxel_host->orient);
                    voxel_remote->changeOrientationTo(voxel_host->orient);
                    voxel_remote->links[voxel_remote_direction]->detach();
                } else if (voxel_host_smallbar) { // host is a small bar
                    voxel_host->adjacentVoxel((linkDirection)voxel_host_direction)->changeOrientationTo(voxel_remote->orient);
                    voxel_host->changeOrientationTo(voxel_remote->orient);
                    voxel_host->links[voxel_host_direction]->detach();
                }
                atomicExch(&d_kernel->mutexRotateSingleton, 0);
                hasSingleton = true;
            }
        }
    }

    if (relativeRotation.w > 0.866 || hasSingleton) // within 30 degree
    {
        VX3_Vec3D<> raw_pos = voxel_remote->position() - voxel_host->position();
        VX3_Vec3D<> pos = voxel_host->orientation().RotateVec3DInv(raw_pos); // the position of remote voxel relative to host voxel.
        if (abs(pos.x) > abs(pos.y) && abs(pos.x) > abs(pos.z)) {
            potential_link_1 = (int)(pos.x > 0 ? X_POS : X_NEG);
            offset_of_link.x = pos.x > 0 ? 1 : -1;
        } else if (abs(pos.y) > abs(pos.z)) {
            potential_link_1 = (int)(pos.y > 0 ? Y_POS : Y_NEG);
            offset_of_link.y = pos.y > 0 ? 1 : -1;
        } else {
            potential_link_1 = (int)(pos.z > 0 ? Z_POS : Z_NEG);
            offset_of_link.z = pos.z > 0 ? 1 : -1;
        }
        potential_link_2 = oppositeDirection(potential_link_1); // only support oppositeDirection attachment for now. Arbitrary attachment is much more difficult.
    } else {
        // Too large of an angle
        return false;
    }

    if (hasSingleton) { // no need to use group map to check for singletons
        if (voxel_host->links[potential_link_1] == NULL and voxel_remote->links[potential_link_2] == NULL) {
            *ret_linkdir_1 = potential_link_1;
            *ret_linkdir_2 = potential_link_2;
            return true;
        } else {
            // DEBUG_PRINT("%f) BAD position for the rotated singleton. The singleton might in between two linked voxels. Skip.\n", d_kernel->currentTime);
            return false;
        }
    }

    // Start checking for compatibility.
    // e.g. a 2D example:
    //     Host: 0,0 - 1,0 - 2,0
    //                  ?
    //     Remote:     0,1
    //                  |
    //                 0,0
    // potential_link_1 = Y_NEG
    // position_of_remote_voxel_in_host_group = Voxel(in remote group).groupPosition + remote_diff;
    // remote_diff = Host.groupPosition + offset_of_link - Remote.groupPosition
    // so (0,0) in Remote group becomes (1,-2).
    bool ret = true;
    VX3_VoxelGroup *remote_group = voxel_remote->d_group;
    VX3_Vec3D<int> remote_diff = VX3_Vec3D<int>(0, 0, 0);
    remote_diff = voxel_host->groupPosition + offset_of_link - voxel_remote->groupPosition;
    for (int i = 0; i < remote_group->d_surface_voxels.size(); i++) {
        VX3_Voxel *v = remote_group->d_surface_voxels[i];
        VX3_Vec3D<int> position_of_remote_voxel_in_host_group = v->groupPosition + remote_diff;
        int offset = to1D(position_of_remote_voxel_in_host_group, VX3_Vec3D<int>(dim_x, dim_y, dim_z));
        if (offset == -1) {
            // good, because out of range
        } else if (d_group_map[offset] == NULL) {
            // good, because empty position
        } else {
            // printf("Not Compatible. Offset %d\n", offset); // Instead of return false, absorb the voxel!
            // TODO: Sida: Is this still in-place modifications?
            VX3_Voxel *voxel_to_absorb = d_group_map[offset];
            if (voxel_to_absorb == voxel_host) {
                ret = false; // Sida: this is a weird situation, the collision happened, but before this check, another voxel has been attached to this exact position. so this collision should not cause attachment.
            } else {
                voxel_to_absorb->removed = true;
                // delete all the links as well
                for (int i = 0; i < 6; i++) {
                    if (voxel_to_absorb->links[i]) {
                        voxel_to_absorb->links[i]->removed = true;
                        voxel_to_absorb->adjacentVoxel((linkDirection)i)->links[oppositeDirection(i)] = NULL;
                        voxel_to_absorb->links[i] = NULL;
                    }
                }
            }
        }
    }
    *ret_linkdir_1 = potential_link_1;
    *ret_linkdir_2 = potential_link_2;
    return ret;
}
